#include "hip/hip_runtime.h"
#include "SPHSystemCUDA.cuh"
#include <hip/hip_runtime.h>
#include <stdexcept>
#include <cstdlib>
#include <ctime>

// ---------- device‑side math helpers (same formulas as CPU) ------------------
__device__ float poly6Kernel(float r2, float h) {
    float h2 = h*h;
    if (r2 >= h2) return 0.f;
    float diff = h2 - r2;
    float coeff = 315.f / (64.f * M_PI * powf(h, 9));
    return coeff * diff * diff * diff;
}
__device__ float viscosityLaplacian(float r, float h){
    return (r>=h)?0.f : 45.f/(M_PI*powf(h,6))*(h-r);
}
__device__ float3 spikyGradient(float3 rij,float h){
    float r = sqrtf(rij.x*rij.x + rij.y*rij.y + rij.z*rij.z);
    if (r==0.f || r>h) return make_float3(0,0,0);
    float coeff = -45.f/(M_PI*powf(h,6))*powf(h-r,2)/r;
    return make_float3(coeff*rij.x, coeff*rij.y, coeff*rij.z);
}

// ---------- kernels ----------------------------------------------------------
__global__ void densityPressureKernel(
        int N, const float3* pos, float* dens, float* pres)
{
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i>=N) return;

    float density = 0.f;
    float3 pi = pos[i];

    for (int j=0; j<N; ++j) {
        float3 rij{
            pos[j].x - pi.x,
            pos[j].y - pi.y,
            pos[j].z - pi.z};
        float r2 = rij.x*rij.x + rij.y*rij.y + rij.z*rij.z;
        density += MASS * poly6Kernel(r2, SMOOTHING_RADIUS);
    }
    dens[i] = density;
    pres[i] = GAS_CONSTANT * (density - REST_DENSITY);
}

__global__ void forceKernel(
        int N,const float3* pos,const float3* vel,
        const float* dens,const float* pres,
        float3* force)
{
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i>=N) return;

    float3 fi = make_float3(0,0,0);
    float3 pi = pos[i];
    float3 vi = vel[i];
    float  di = dens[i];
    float  pi_pres = pres[i];

    for (int j=0;j<N;++j){
        if (i==j) continue;
        float3 rij{
            pi.x - pos[j].x,
            pi.y - pos[j].y,
            pi.z - pos[j].z};
        float  r  = sqrtf(rij.x*rij.x + rij.y*rij.y + rij.z*rij.z);

        // pressure
        float3 grad = spikyGradient(rij, SMOOTHING_RADIUS);
        fi.x += -MASS*(pi_pres+pres[j])/(2.f*dens[j])*grad.x;
        fi.y += -MASS*(pi_pres+pres[j])/(2.f*dens[j])*grad.y;
        fi.z += -MASS*(pi_pres+pres[j])/(2.f*dens[j])*grad.z;

        // viscosity
        float lap = viscosityLaplacian(r, SMOOTHING_RADIUS);
        fi.x += VISCOSITY*MASS*(vel[j].x-vi.x)/dens[j]*lap;
        fi.y += VISCOSITY*MASS*(vel[j].y-vi.y)/dens[j]*lap;
        fi.z += VISCOSITY*MASS*(vel[j].z-vi.z)/dens[j]*lap;
    }
    // gravity
    fi.y += di*GRAVITY;

    force[i]=fi;
}

__global__ void integrateKernel(
        int N,float3* pos,float3* vel,const float3* force,const float* dens)
{
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i>=N) return;

    // semi‑implicit Euler
    float3 v = vel[i];
    float3 f = force[i];
    float  invMass = 1.f/dens[i];   // actually 1/ρ not mass
    v.x += f.x*invMass*TIME_STEP;
    v.y += f.y*invMass*TIME_STEP;
    v.z += f.z*invMass*TIME_STEP;

    float3 p = pos[i];
    p.x += v.x*TIME_STEP;
    p.y += v.y*TIME_STEP;
    p.z += v.z*TIME_STEP;

    // bounding box [0,1]×[0,0.5]×[0,1]
    const float3 bmin{0,0,0}, bmax{1,0.5,1};
    if(p.x<bmin.x){p.x=bmin.x; v.x*=DAMPING;}
    if(p.x>bmax.x){p.x=bmax.x; v.x*=DAMPING;}
    if(p.y<bmin.y){p.y=bmin.y; v.y*=DAMPING;}
    if(p.y>bmax.y){p.y=bmax.y; v.y*=DAMPING;}
    if(p.z<bmin.z){p.z=bmin.z; v.z*=DAMPING;}
    if(p.z>bmax.z){p.z=bmax.z; v.z*=DAMPING;}

    pos[i]=p; vel[i]=v;
}

// ---------- host‑side constructor / destructor ------------------------------
SPHSystemCUDA::SPHSystemCUDA()
{
    initializeParticles();
    N_ = static_cast<int>(particles.size());

    // allocate & copy to device
    size_t v3 = N_*sizeof(float3), v1 = N_*sizeof(float);
    hipMalloc(&d_pos_, v3);
    hipMalloc(&d_vel_, v3);
    hipMalloc(&d_force_, v3);
    hipMalloc(&d_density_, v1);
    hipMalloc(&d_pressure_, v1);

    std::vector<float3> h_pos(N_), h_vel(N_);
    for(int i=0;i<N_;++i){
        h_pos[i] = make_float3(particles[i].position.x,
                               particles[i].position.y,
                               particles[i].position.z);
        h_vel[i] = make_float3(0,0,0);
    }
    hipMemcpy(d_pos_, h_pos.data(), v3, hipMemcpyHostToDevice);
    hipMemcpy(d_vel_, h_vel.data(), v3, hipMemcpyHostToDevice);
}

SPHSystemCUDA::~SPHSystemCUDA(){
    hipFree(d_pos_); hipFree(d_vel_); hipFree(d_force_);
    hipFree(d_density_); hipFree(d_pressure_);
}

// Initialize particles in a noisy grid (mirroring CPU version)
void SPHSystemCUDA::initializeParticles() {
    std::srand(static_cast<unsigned>(std::time(nullptr)));
    float noiseScale = spacing * 0.1f;
    for (int x = 0; x < numX; ++x) {
        for (int y = 0; y < numY; ++y) {
            for (int z = 0; z < numZ; ++z) {
                float nx = ((std::rand() % 1000) / 1000.0f - 0.5f) * noiseScale;
                float ny = ((std::rand() % 1000) / 1000.0f - 0.5f) * noiseScale;
                float nz = ((std::rand() % 1000) / 1000.0f - 0.5f) * noiseScale;
                glm::vec3 pos(
                    x * spacing + nx,
                    y * spacing + 0.5f + ny,
                    z * spacing + nz
                );
                particles.emplace_back(pos);
            }
        }
    }
}

// ---------- public API: launch kernels & copy back for renderer -------------
inline dim3 gridFor(int N,int block){ return dim3((N+block-1)/block); }

void SPHSystemCUDA::computeDensityPressure(){
    densityPressureKernel<<<gridFor(N_,256),256>>>(N_,d_pos_,d_density_,d_pressure_);
    hipDeviceSynchronize();
}

void SPHSystemCUDA::computeForces(){
    forceKernel<<<gridFor(N_,256),256>>>(N_,d_pos_,d_vel_,d_density_,d_pressure_,d_force_);
    hipDeviceSynchronize();
}

void SPHSystemCUDA::integrate(){
    integrateKernel<<<gridFor(N_,256),256>>>(N_,d_pos_,d_vel_,d_force_,d_density_);
    hipDeviceSynchronize();

    // copy positions back so ParticleRenderer can update VBO
    std::vector<float3> h_pos(N_);
    hipMemcpy(h_pos.data(), d_pos_, N_*sizeof(float3), hipMemcpyDeviceToHost);

    for (int i=0;i<N_;++i){
        particles[i].position = glm::vec3(h_pos[i].x,h_pos[i].y,h_pos[i].z);
    }
}

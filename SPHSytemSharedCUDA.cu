#include "hip/hip_runtime.h"
// conclusion: the shared memory approach doesn't make improvement compared to global memory approach. It could be flawed code or the nature of SPH.

#include "SPHSystemCUDA.cuh"
#include <hip/hip_runtime.h>
#include <stdexcept>
#include <cstdlib>
#include <ctime>

// ---------- device‑side math helpers (same formulas as CPU) ------------------
__device__ float poly6Kernel(float r2, float h) {
    float h2 = h*h;
    if (r2 >= h2) return 0.f;
    float diff = h2 - r2;
    float coeff = 315.f / (64.f * M_PI * powf(h, 9));
    return coeff * diff * diff * diff;
}
__device__ float viscosityLaplacian(float r, float h){
    return (r>=h)?0.f : 45.f/(M_PI*powf(h,6))*(h-r);
}
__device__ float3 spikyGradient(float3 rij,float h){
    float r = sqrtf(rij.x*rij.x + rij.y*rij.y + rij.z*rij.z);
    if (r==0.f || r>h) return make_float3(0,0,0);
    float coeff = -45.f/(M_PI*powf(h,6))*powf(h-r,2)/r;
    return make_float3(coeff*rij.x, coeff*rij.y, coeff*rij.z);
}

// ---------- kernels ----------------------------------------------------------

__global__ void densityPressureKernel(
    int N, const float3* pos, float* dens, float* pres)
{
int id = blockIdx.x*blockDim.x + threadIdx.x; //block_id * num_thread_per_block + thread_id

if (id>=N) return;

float density = 0.f;
float3 pi = pos[id];

for (int j=0; j<N; ++j) {
    float3 rij{
        pos[j].x - pi.x,
        pos[j].y - pi.y,
        pos[j].z - pi.z};
    float r2 = rij.x*rij.x + rij.y*rij.y + rij.z*rij.z;
    density += MASS * poly6Kernel(r2, SMOOTHING_RADIUS);
}
dens[id] = density;
pres[id] = GAS_CONSTANT * (density - REST_DENSITY);
}

__global__ void densityPressureKernel_shared(
    int N, const float3* pos, float* dens, float* pres)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N) return;

    float3 pi = pos[i];
    float density = 0.0f;

    // Shared memory tile
    __shared__ float3 shPos[256]; //CUDA doesn't allow shPos[blockDim.x];

    int tileSize = blockDim.x;  //256
    int numTiles = (N + tileSize - 1) / tileSize; //if #particle is 1,000, #tiles is 4

    for (int tile = 0; tile < numTiles; ++tile) {
        int j = tile * blockDim.x + threadIdx.x; //Thread_id3: 0*256 + 3, 1*256 + 3, 2*256 + 3, 3*256 + 3

        // Load particle j into shared memory
        if (j < N)
            shPos[threadIdx.x] = pos[j]; //overwriting shPos? 

        __syncthreads(); //avoid race condition: Some threads may start reading shPos[k] before others have finished writing to it
        //at this point, shPos will be filled with all particles
        
        // Iterate over particles within the current tile
        for (int k = 0; k < blockDim.x; ++k) {
            int actualIdx = tile * blockDim.x + k;
            if (actualIdx >= N) break;

            float3 rij;
            rij.x = shPos[k].x - pi.x;
            rij.y = shPos[k].y - pi.y;
            rij.z = shPos[k].z - pi.z;

            float r2 = rij.x * rij.x + rij.y * rij.y + rij.z * rij.z;
            density += MASS * poly6Kernel(r2, SMOOTHING_RADIUS);
        }

        __syncthreads(); //avoid race condition: prevent threads from overwriting shPos
    }

    dens[i] = density;
    pres[i] = GAS_CONSTANT * (density - REST_DENSITY);
}

//this kernel doesn't work as expected. 
// 1) can see particles in the very first frame but then disappear 
// 2) the value shDens NaN, which means not loaded correctly. 
// 3) when removing shDens, particles move but velocity looks different. 
__global__ void forceKernel_shared(
    int N,
    const float3* pos, const float3* vel,
    const float* dens, const float* pres,
    float3* force)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N) return;

    float3 fi = make_float3(0, 0, 0);
    float3 pi = pos[i];
    float3 vi = vel[i];
    float  di = dens[i];
    float  pi_pres = pres[i];

    // Shared memory
    __shared__ float3 shPos[256];
    __shared__ float3 shVel[256];
    __shared__ float  shDens[256];
    __shared__ float  shPres[256];

    int tileSize = blockDim.x;
    int numTiles = (N + tileSize - 1) / tileSize;

    for (int tile = 0; tile < numTiles; ++tile) {
        int j = tile * tileSize + threadIdx.x;

        if (j < N) {
            shPos[threadIdx.x]  = pos[j];
            shVel[threadIdx.x]  = vel[j];
            shDens[threadIdx.x] = dens[j];
            shPres[threadIdx.x] = pres[j];
        }

        __syncthreads();

        for (int k = 0; k < tileSize; ++k) {
            int jIdx = tile * tileSize + k;
            if (jIdx >= N || jIdx == i) continue;

            float3 rij = {
                pi.x - shPos[k].x,
                pi.y - shPos[k].y,
                pi.z - shPos[k].z
            };

            float r2 = rij.x * rij.x + rij.y * rij.y + rij.z * rij.z;
            float r  = sqrtf(fmaxf(r2, 1e-12f));  // avoid sqrt(0)

            // Pressure force
            float3 grad = spikyGradient(rij, SMOOTHING_RADIUS);
            // float pressureScale = -MASS * (pi_pres + shPres[k]) / (2.0f * dens[jIdx]);
            float pressureScale = -MASS * (pi_pres + shPres[k]) / (2.0f * shDens[k]);
            fi.x += pressureScale * grad.x;
            fi.y += pressureScale * grad.y;
            fi.z += pressureScale * grad.z;

            // Viscosity force
            float lap = viscosityLaplacian(r, SMOOTHING_RADIUS);
            float3 vDiff = {
                shVel[k].x - vi.x,
                shVel[k].y - vi.y,
                shVel[k].z - vi.z
            };
            float viscosityScale = VISCOSITY * MASS / shDens[k];
            // float viscosityScale = VISCOSITY * MASS / dens[jIdx];
            fi.x += viscosityScale * vDiff.x * lap;
            fi.y += viscosityScale * vDiff.y * lap;
            fi.z += viscosityScale * vDiff.z * lap;
        }

        __syncthreads();  // ensure reads are done before loading next tile
    }

    // Gravity
    fi.y += di * GRAVITY;
    force[i] = fi;
}

__global__ void forceKernel(
    int N,const float3* pos,const float3* vel,
    const float* dens,const float* pres,
    float3* force)
{
int i = blockIdx.x*blockDim.x + threadIdx.x;
if (i>=N) return;

float3 fi = make_float3(0,0,0);
float3 pi = pos[i];
float3 vi = vel[i];
float  di = dens[i];
float  pi_pres = pres[i];

for (int j=0;j<N;++j){
    if (i==j) continue;
    float3 rij{
        pi.x - pos[j].x,
        pi.y - pos[j].y,
        pi.z - pos[j].z};
    float  r  = sqrtf(rij.x*rij.x + rij.y*rij.y + rij.z*rij.z);

    // pressure
    float3 grad = spikyGradient(rij, SMOOTHING_RADIUS);
    fi.x += -MASS*(pi_pres+pres[j])/(2.f*dens[j])*grad.x;
    fi.y += -MASS*(pi_pres+pres[j])/(2.f*dens[j])*grad.y;
    fi.z += -MASS*(pi_pres+pres[j])/(2.f*dens[j])*grad.z;

    // viscosity
    float lap = viscosityLaplacian(r, SMOOTHING_RADIUS);
    fi.x += VISCOSITY*MASS*(vel[j].x-vi.x)/dens[j]*lap;
    fi.y += VISCOSITY*MASS*(vel[j].y-vi.y)/dens[j]*lap;
    fi.z += VISCOSITY*MASS*(vel[j].z-vi.z)/dens[j]*lap;
}
// gravity
fi.y += di*GRAVITY;

force[i]=fi;
}

__global__ void integrateKernel_shared(
    int N, float3* pos, float3* vel,
    const float3* force, const float* dens)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N) return;

    // Fixed-size shared memory for current thread's force and density
    __shared__ float3 shForce[256];
    __shared__ float  shDens[256];

    // Load per-thread values into shared memory
    shForce[threadIdx.x] = force[i];
    shDens[threadIdx.x] = dens[i];

    __syncthreads();  // ensure values are available to threadIdx.x

    float3 v = vel[i];
    float3 f = shForce[threadIdx.x];
    float  invMass = 1.f / shDens[threadIdx.x];

    // Semi-implicit Euler integration
    v.x += f.x * invMass * TIME_STEP;
    v.y += f.y * invMass * TIME_STEP;
    v.z += f.z * invMass * TIME_STEP;

    float3 p = pos[i];
    p.x += v.x * TIME_STEP;
    p.y += v.y * TIME_STEP;
    p.z += v.z * TIME_STEP;

    // Bounding box [0,1]×[0,0.5]×[0,1]
    const float3 bmin{0, 0, 0}, bmax{1, 0.5, 1};
    if (p.x < bmin.x) { p.x = bmin.x; v.x *= DAMPING; }
    if (p.x > bmax.x) { p.x = bmax.x; v.x *= DAMPING; }
    if (p.y < bmin.y) { p.y = bmin.y; v.y *= DAMPING; }
    if (p.y > bmax.y) { p.y = bmax.y; v.y *= DAMPING; }
    if (p.z < bmin.z) { p.z = bmin.z; v.z *= DAMPING; }
    if (p.z > bmax.z) { p.z = bmax.z; v.z *= DAMPING; }

    pos[i] = p;
    vel[i] = v;
}


__global__ void integrateKernel(
        int N,float3* pos,float3* vel,const float3* force,const float* dens)
{
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i>=N) return;

    // semi‑implicit Euler
    float3 v = vel[i];
    float3 f = force[i];
    float  invMass = 1.f/dens[i];   // actually 1/ρ not mass
    v.x += f.x*invMass*TIME_STEP;
    v.y += f.y*invMass*TIME_STEP;
    v.z += f.z*invMass*TIME_STEP;

    float3 p = pos[i];
    p.x += v.x*TIME_STEP;
    p.y += v.y*TIME_STEP;
    p.z += v.z*TIME_STEP;

    // bounding box [0,1]×[0,0.5]×[0,1]
    const float3 bmin{0,0,0}, bmax{1,0.5,1};
    if(p.x<bmin.x){p.x=bmin.x; v.x*=DAMPING;}
    if(p.x>bmax.x){p.x=bmax.x; v.x*=DAMPING;}
    if(p.y<bmin.y){p.y=bmin.y; v.y*=DAMPING;}
    if(p.y>bmax.y){p.y=bmax.y; v.y*=DAMPING;}
    if(p.z<bmin.z){p.z=bmin.z; v.z*=DAMPING;}
    if(p.z>bmax.z){p.z=bmax.z; v.z*=DAMPING;}

    pos[i]=p; vel[i]=v;
}

// ---------- host‑side constructor / destructor ------------------------------
SPHSystemCUDA::SPHSystemCUDA()
{
    initializeParticles();
    N_ = static_cast<int>(particles.size());

    // allocate & copy to device
    size_t v3 = N_*sizeof(float3), v1 = N_*sizeof(float);
    hipMalloc(&d_pos_, v3);
    hipMalloc(&d_vel_, v3);
    hipMalloc(&d_force_, v3);
    hipMalloc(&d_density_, v1);
    hipMalloc(&d_pressure_, v1);

    std::vector<float3> h_pos(N_), h_vel(N_);
    for(int i=0;i<N_;++i){
        h_pos[i] = make_float3(particles[i].position.x,
                               particles[i].position.y,
                               particles[i].position.z);
        h_vel[i] = make_float3(0,0,0);
    }
    hipMemcpy(d_pos_, h_pos.data(), v3, hipMemcpyHostToDevice);
    hipMemcpy(d_vel_, h_vel.data(), v3, hipMemcpyHostToDevice);
}

SPHSystemCUDA::~SPHSystemCUDA(){
    hipFree(d_pos_); hipFree(d_vel_); hipFree(d_force_);
    hipFree(d_density_); hipFree(d_pressure_);
}

// Initialize particles in a noisy grid (mirroring CPU version)
void SPHSystemCUDA::initializeParticles() {
    std::srand(static_cast<unsigned>(std::time(nullptr)));
    float noiseScale = spacing * 0.1f;
    for (int x = 0; x < numX; ++x) {
        for (int y = 0; y < numY; ++y) {
            for (int z = 0; z < numZ; ++z) {
                float nx = ((std::rand() % 1000) / 1000.0f - 0.5f) * noiseScale;
                float ny = ((std::rand() % 1000) / 1000.0f - 0.5f) * noiseScale;
                float nz = ((std::rand() % 1000) / 1000.0f - 0.5f) * noiseScale;
                glm::vec3 pos(
                    x * spacing + nx,
                    y * spacing + 0.5f + ny,
                    z * spacing + nz
                );
                particles.emplace_back(pos);
            }
        }
    }
}

//helper function designed to calculate the necessary grid dimension (gridDim)
inline dim3 gridFor(int N,int block){ return dim3((N+block-1)/block); }

void SPHSystemCUDA::computeDensityPressure(){
    densityPressureKernel_shared<<<gridFor(N_,256),256>>>(N_,d_pos_,d_density_,d_pressure_);
    hipDeviceSynchronize();
}

void SPHSystemCUDA::computeForces(){
    forceKernel<<<gridFor(N_,256),256>>>(N_,d_pos_,d_vel_,d_density_,d_pressure_,d_force_);
    hipDeviceSynchronize();
}

void SPHSystemCUDA::integrate(){
    integrateKernel<<<gridFor(N_,256),256>>>(N_,d_pos_,d_vel_,d_force_,d_density_);
    hipDeviceSynchronize();

    // copy positions back so ParticleRenderer can update VBO
    std::vector<float3> h_pos(N_);
    hipMemcpy(h_pos.data(), d_pos_, N_*sizeof(float3), hipMemcpyDeviceToHost);

    for (int i=0;i<N_;++i){
        particles[i].position = glm::vec3(h_pos[i].x,h_pos[i].y,h_pos[i].z);
    }
}
